#include "hip/hip_runtime.h"
#include <iostream>
#include <sys/time.h>

#define min(a, b) (a < b ? a : b)

__device__ unsigned int getIdx(dim3* threads, dim3* blocks) {
    int x;
    return threadIdx.x +
           threadIdx.y * (x  = threads->x) +
           threadIdx.z * (x *= threads->y) +
           blockIdx.x  * (x *= threads->z) +
           blockIdx.y  * (x *= blocks->z) +
           blockIdx.z  * (x *= blocks->y);
}

__device__ void gpu_bottomUpMerge(double* source, double* dest, long start, long middle, long end) {
    long i = start;
    long j = middle;
    for (long k = start; k < end; k++) {
        if (i < middle && (j >= end || source[i] < source[j])) {
            dest[k] = source[i];
            i++;
        } else {
            dest[k] = source[j];
            j++;
        }
    }
}

__global__ void gpu_mergesort(double* source, double* dest, long size, long width, long slices, dim3* threads, dim3* blocks) {
    unsigned int idx = getIdx(threads, blocks);
    long start = width*idx*slices, 
         middle, 
         end;

    for (long slice = 0; slice < slices; slice++) {
        if (start >= size)
            break;

        middle = min(start + (width >> 1), size);
        end = min(start + width, size);
        gpu_bottomUpMerge(source, dest, start, middle, end);
        start += width;
    }
}





int main(int argc, char** argv) {

      double* data;

    long size = 100000;
    data = (double *)malloc(size*sizeof(double));
    for(int i = size - 1; i >= 0; i--) {
	    data[i] = i;
    }

    std::cout << "sorting " << size << " numbers\n\n";

    mergesort_gpu(data, size);


    for (int i = 0; i < size; i++) {
        std::cout << data[i] << '\n';
    } 
}

void mergesort_gpu(double* data, long size, int xThreadPerBlock, int xBlocksPerGrid) {

    dim3 threadsPerBlock;
    dim3 blocksPerGrid;

    threadsPerBlock.x = xThreadPerBlock;
    threadsPerBlock.y = 1;
    threadsPerBlock.z = 1;

    blocksPerGrid.x = xBlocksPerGrid;
    blocksPerGrid.y = 1;
    blocksPerGrid.z = 1;



    double* D_data;
    double* D_swp;
    dim3* D_threads;
    dim3* D_blocks;
    
    hipMalloc((void**) &D_data, size * sizeof(double));
    hipMalloc((void**) &D_swp, size * sizeof(double));

    hipMemcpy(D_data, data, size * sizeof(long), hipMemcpyHostToDevice);
 
    hipMalloc((void**) &D_threads, sizeof(dim3));
    hipMalloc((void**) &D_blocks, sizeof(dim3));

    hipMemcpy(D_threads, &threadsPerBlock, sizeof(dim3), hipMemcpyHostToDevice);
    hipMemcpy(D_blocks, &blocksPerGrid, sizeof(dim3), hipMemcpyHostToDevice);


    double* A = D_data;
    double* B = D_swp;

    long nThreads = threadsPerBlock.x * threadsPerBlock.y * threadsPerBlock.z *
                    blocksPerGrid.x * blocksPerGrid.y * blocksPerGrid.z;

    for (int width = 2; width < (size << 1); width <<= 1) {
        long slices = size / ((nThreads) * width) + 1;

        gpu_mergesort<<<blocksPerGrid, threadsPerBlock>>>(A, B, size, width, slices, D_threads, D_blocks);


        A = A == D_data ? D_swp : D_data;
        B = B == D_data ? D_swp : D_data;
    }

    hipMemcpy(data, A, size * sizeof(double), hipMemcpyDeviceToHost);
    
    
    hipFree(A);
    hipFree(B);
}


